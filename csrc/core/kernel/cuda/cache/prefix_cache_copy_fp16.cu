
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) Alibaba, Inc. and its affiliates.
 * @file    prefix_cache_copy_fp16.cu
 */

#ifdef ENABLE_FP16
#include <cuda_fp16.h>

#include "prefix_cache_copy.cuh"

namespace allspark {
namespace cuda {

template void PrefixCacheCopyLauncher(const void* const* spanPtrs, half* dst,
                                      int nGroups, int headSize, int spanLen,
                                      int preLen, span::QuantMode cacheMode,
                                      cudaStream_t stream);

}  // namespace cuda
}  // namespace allspark
#endif
