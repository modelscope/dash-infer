/*!
 * Copyright (c) Alibaba, Inc. and its affiliates.
 * @file    decoder_cache_append_fp32.cu
 */

#include "decoder_cache_append.cuh"

namespace allspark {
namespace cuda {

template void DecoderCacheAppendLauncher(
    void* const* kSpanArray, void* const* vSpanArray, float* queryOut,
    const float* src, const uint32_t* oldSeqLens, int batchSize, int nHeads,
    int nGroups, int headSize, int spanLen, int nSpansPerBatch,
    span::QuantMode cacheMode, hipStream_t stream);

}  // namespace cuda
}  // namespace allspark
