
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) Alibaba, Inc. and its affiliates.
 * @file    bf16_simt.cu
 */

#ifdef ENABLE_BF16
#include "attn/qk/impl_simt.cuh"
#include "common/data_type.h"

namespace span {
template struct QKLauncher<SaArch::SIMT, bfloat16_t>;
}  // namespace span
#endif
