#include "hip/hip_runtime.h"
/*!
 * Copyright (c) Alibaba, Inc. and its affiliates.
 * @file    hiednn_cuda.cu
 */

#include <hiednn.h>
#include <hiednn_cuda.h>
#include <cuda/cuda_utils.hpp>
#include <cuda/cuda_handle.hpp>

hiednnStatus_t
hiednnCreateCudaHandle(HiednnCudaHandle **handle) {
    *handle = new HiednnCudaHandle();
    HiednnCudaHandle &handleRef = **handle;

    // cuda runtime initialization
    CHECK_CUDA_RETURN(hipFree(0));

    // device property
    int deviceId;
    CHECK_CUDA_RETURN(hipGetDevice(&deviceId));
    CHECK_CUDA_RETURN(hipGetDeviceProperties(&handleRef.deviceProp, deviceId));
    size_t totalGlobalMem = handleRef.deviceProp.totalGlobalMem;

    // default stream
    handleRef.stream = 0;

    // allocate cuda memory workspace
    const size_t giga = (1LU << 30);
    const size_t mega = (1LU << 20);

    if (totalGlobalMem >= 31 * giga) {
        // 512MB workspace for GPUs with at least 32GB memory
        handleRef.deviceWsSize = 512 * mega;
    } else if (totalGlobalMem >= 23 * giga) {
        // 256MB workspace for GPUs with at least 24GB memory
        handleRef.deviceWsSize = 256 * mega;
    } else if (totalGlobalMem >= 7 * giga) {
        // 128MB workspace for GPUs with at least 8GB memory
        handleRef.deviceWsSize = 128 * mega;
    } else {
        // 64MB workspace for GPUs with less tan 8GB memory
        handleRef.deviceWsSize = 64 * mega;
    }

    CHECK_CUDA_RETURN(hipMalloc(
        &handleRef.deviceWs, handleRef.deviceWsSize));
    CHECK_CUDA_RETURN(hipEventCreateWithFlags(
        &handleRef.wsMutex, hipEventDisableTiming));

    return HIEDNN_STATUS_SUCCESS;
}

hiednnStatus_t
hiednnDestroyCudaHandle(HiednnCudaHandle *handle) {
    if (handle != nullptr) {
        // release cuda memory workspace
        auto err = hipFree(handle->deviceWs);
        if (err != hipSuccess && err != hipErrorDeinitialized) {
            return HIEDNN_STATUS_RUNTIME_ERROR;
        }

        err = hipEventDestroy(handle->wsMutex);
        if (err != hipSuccess && err != hipErrorDeinitialized) {
            return HIEDNN_STATUS_RUNTIME_ERROR;
        }

        delete handle;
        return HIEDNN_STATUS_SUCCESS;
    } else {
        return HIEDNN_STATUS_INVALID_PARAMETER;
    }
}

hiednnStatus_t
hiednnSetCudaStream(HiednnCudaHandle *handle, hipStream_t stream) {
    if (handle != nullptr) {
        handle->stream = stream;
        return HIEDNN_STATUS_SUCCESS;
    } else {
        return HIEDNN_STATUS_INVALID_PARAMETER;
    }
}


