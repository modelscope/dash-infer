#include "hip/hip_runtime.h"
/*!
 * Copyright (c) Alibaba, Inc. and its affiliates.
 * @file    10_linear_interpolation.cu
 */

#include <hiednn.h>
#include <hiednn_cuda.h>

#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cmath>

#define CHECK_CUDA(expr) { \
    if ((expr) != hipSuccess) { \
        int line = __LINE__; \
        printf("cuda error at %d\n", line); \
        exit(1); \
    } \
}

#define CHECK_HIEDNN(expr) { \
    if ((expr) != HIEDNN_STATUS_SUCCESS) { \
        int line = __LINE__; \
        printf("hiednn error at %d\n", line); \
        exit(1); \
    } \
}

#define CHECK_32F(y, ref) { \
    if (std::fabs(y - ref) > std::fabs(ref) * 1e-5f) { \
        printf("FAILED\n"); \
        exit(1); \
    } \
}

float Asymmetric(float yCoord, float scale) {
    return yCoord / scale;
}

void LinearInterp2DReference(
        const float *x, const int64_t *xDim, const float *scale,
        float *y, const int64_t *yDim) {
    for (int64_t batch = 0; batch < yDim[0]; ++batch) {
        const float *xPtr = x + batch * xDim[1] * xDim[2];
        float *yPtr = y + batch * yDim[1] * yDim[2];

        for (int64_t row = 0; row < yDim[1]; ++row) {
            for (int64_t col = 0; col < yDim[2]; ++col) {
                float xCoordF[2];
                xCoordF[0] = Asymmetric(static_cast<float>(row), scale[0]);
                xCoordF[1] = Asymmetric(static_cast<float>(col), scale[1]);

                float xCoordFL[2];
                float xCoordFR[2];
                xCoordFL[0] = std::floor(xCoordF[0]);
                xCoordFL[1] = std::floor(xCoordF[1]);
                xCoordFR[0] = std::ceil(xCoordF[0]);
                xCoordFR[1] = std::ceil(xCoordF[1]);
                if (xCoordFR[0] >= xDim[1]) xCoordFR[0] = xDim[1] - 1;
                if (xCoordFR[1] >= xDim[2]) xCoordFR[1] = xDim[2] - 1;

                int64_t xCoordIL[2];
                int64_t xCoordIR[2];
                xCoordIL[0] = static_cast<int64_t>(xCoordFL[0]);
                xCoordIL[1] = static_cast<int64_t>(xCoordFL[1]);
                xCoordIR[0] = static_cast<int64_t>(xCoordFR[0]);
                xCoordIR[1] = static_cast<int64_t>(xCoordFR[1]);

                float xCoordOffset[2];
                xCoordOffset[0] = xCoordF[0] - xCoordFL[0];
                xCoordOffset[1] = xCoordF[1] - xCoordFL[1];

                int64_t xOffset00 = xCoordIL[0] * xDim[2] + xCoordIL[1];
                int64_t xOffset01 = xCoordIL[0] * xDim[2] + xCoordIR[1];
                int64_t xOffset10 = xCoordIR[0] * xDim[2] + xCoordIL[1];
                int64_t xOffset11 = xCoordIR[0] * xDim[2] + xCoordIR[1];

                float x00 = xPtr[xOffset00];
                float x01 = xPtr[xOffset01];
                float x10 = xPtr[xOffset10];
                float x11 = xPtr[xOffset11];

                float x0 = x00 + (x01 - x00) * xCoordOffset[1];
                float x1 = x10 + (x11 - x10) * xCoordOffset[1];

                float yRef = x0 + (x1 - x0) * xCoordOffset[0];
                yPtr[row * yDim[2] + col] = yRef;
            }
        }
    }
}

int main() {
    /*
     * input:
     * tensor x: dim={2, 4, 4}, dataType=float
     * param coorMode=HIEDNN_INTERP_COORD_ASYMMETRIC
     * param scale={1.5, 1.5}
     *
     * output:
     * tensor y: dim={2, 6. 6}, datatype=float
     */
    // input dimension:
    int64_t xDim[] = {2, 4, 4};
    int64_t xSize = 2 * 4 * 4;
    int xNDims = 3;

    // parameter
    hiednnDataType_t dataType = HIEDNN_DATATYPE_FP32;
    hiednnInterpCoordMode_t coordMode = HIEDNN_INTERP_COORD_ASYMMETRIC;
    float scale[] = {1.5, 1.5};
    int scaleSize = 2;

    // output dimension:
    int64_t yDim[] = {2, 6, 6};
    int64_t ySize = 2 * 6 * 6;
    int yNDims = 3;

    // init tensor x
    float x[2 * 4 * 4];
    for (int i = 0; i < 2 * 4 * 4; ++i) {
        x[i] = 0.5f * i;
    }

    // create cuda handle
    hiednnCudaHandle_t handle;
    CHECK_HIEDNN(hiednnCreateCudaHandle(&handle));

    // default stream
    CHECK_HIEDNN(hiednnSetCudaStream(handle, 0));

    // create tensor descriptor
    hiednnTensorDesc_t xDesc, yDesc;
    CHECK_HIEDNN(hiednnCreateTensorDesc(&xDesc));
    CHECK_HIEDNN(hiednnCreateTensorDesc(&yDesc));

    // init tensor descriptor
    CHECK_HIEDNN(hiednnSetNormalTensorDesc(xDesc, dataType, xNDims, xDim));
    CHECK_HIEDNN(hiednnSetNormalTensorDesc(yDesc, dataType, yNDims, yDim));

    // allocate device memory for tensor and copy input tensor to device
    float *dx, *dy;
    CHECK_CUDA(hipMalloc(&dx, xSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&dy, ySize * sizeof(float)));
    CHECK_CUDA(hipMemcpy(dx, x, xSize * sizeof(float),
                          hipMemcpyHostToDevice));

    CHECK_HIEDNN(hiednnCudaLinearInterpolation(
        handle, coordMode, xDesc, dx, scale, scaleSize, yDesc, dy));

    float *y = static_cast<float *>(malloc(ySize * sizeof(float)));

    // copy output tensor from device to host
    CHECK_CUDA(hipMemcpy(y, dy, ySize * sizeof(float),
                          hipMemcpyDeviceToHost));

    // get expected output
    float *yRef = static_cast<float *>(malloc(ySize * sizeof(float)));
    LinearInterp2DReference(x, xDim, scale, yRef, yDim);

    // check output
    printf("check output tenosr... ");
    for (int i = 0; i < ySize; ++i) {
        CHECK_32F(y[i], yRef[i]);
    }
    printf("OK\n");

    CHECK_HIEDNN(hiednnDestroyTensorDesc(xDesc));
    CHECK_HIEDNN(hiednnDestroyTensorDesc(yDesc));
    CHECK_HIEDNN(hiednnDestroyCudaHandle(handle));

    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(dy));

    free(y);
    free(yRef);

    return 0;
}

